#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#define N 64
#define TPB 32

float scale(int i, int n)
{
	return ((float)i / (n - 1));
}

__device__ 
float distance(float x1, float x2)
{
	return sqrt((x2 - x1) * (x2 - x1) );
}

__global__
void distanceKernel(float* d_out, float ref, float* d_in)
{
	const int i = threadIdx.x + blockIdx.x * blockDim.x;
	const float x = d_in[i];
	d_out[i] = distance(x, ref);
	printf("i = %2d: dist from %f to %f is %f.\n", i, ref, x, d_out[i]);
}

int main()
{
	const float ref = 0.5f;

	float* in = 0;
	float* out = 0;

	// allocate managed memory for input and output
	hipMallocManaged(&in, N * sizeof(float));
	hipMallocManaged(&out, N * sizeof(float));

	// Computer scaled input values
	for (int i = 0; i < N; i++)
	{
		in[i] = scale(i, N);
	}

	// launch kernel
	
 	distanceKernel << <N/TPB, TPB >> > (out, ref, in);
	
	// wait for GPU to finish before accessing on host
	hipDeviceSynchronize();
	
	// free memory
	hipFree(in);
	hipFree(out);
	
	return 0;
}
